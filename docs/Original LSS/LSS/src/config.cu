#include "config.h"

#if defined(CUDA_DEBUG)
__device__ double *debug_buff;
void initialize_debug_buff(size_t size)
{
	double *debug_buff_host;
	hipMalloc(&debug_buff_host, sizeof(double) * size);
	hipMemcpyToSymbol(HIP_SYMBOL(debug_buff), &debug_buff_host, sizeof(double *));
}

void get_debug_buff(void **debug_buff_host, size_t size)
{
	*debug_buff_host = malloc(size);
	double *debug_buff_device;
	hipMemcpyFromSymbol(&debug_buff_device, HIP_SYMBOL(debug_buff), sizeof(double *));
	hipMemcpy(*debug_buff_host, debug_buff_device, size, hipMemcpyDeviceToHost);
}

__device__ void dump(void *data, size_t size)
{
	memcpy(debug_buff, data, size);
}


#endif
