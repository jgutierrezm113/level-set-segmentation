#include "hip/hip_runtime.h"
/* Julian Gutierrez
 * Northeastern University
 * High Performance Computing
 * 
 * Level Set Segmentation for Image Processing 
 *  
 */
 
#include "lss.h"

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != hipSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}

using namespace std;

void modMaxIter (int value){
	max_iterations = value;
}

/*
 * Lss Step 1 from Pseudo Code
 */
__global__ void lssStep1(unsigned int* intensity, 
			 unsigned int* labels,
			 signed int* phi, 
			 int targetLabel, 
			 int lowerIntensityBound, 
			 int upperIntensityBound,
			 int* globalBlockIndicator,
			 int* globalFinishedVariable) {

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	int blockId = by*gridDim.x+bx;
				
	// Including border
	__shared__ signed int intensityTile[TILE_SIZE+2][TILE_SIZE+2]; // input
	__shared__ signed int     labelTile[TILE_SIZE+2][TILE_SIZE+2]; // input
	__shared__ signed int       phiTile[TILE_SIZE+2][TILE_SIZE+2]; // output
	
	// Global Block Indicator
	__shared__ volatile signed int localGBI;
	__shared__ volatile signed int allOnes;
	__shared__ volatile signed int change;
		
	// Read Input Data into Shared Memory
	/////////////////////////////////////////////////////////////////////////////////////

	int x = bx<<BTSB;
	x = x + tx;
	x = x<<TTSB;
	int y = by<<BTSB;
	y = y + ty;
	y = y<<TTSB;
	  
	int location = 	(((x>>TTSB)&BTSMask)                ) |
			(((y>>TTSB)&BTSMask) << BTSB        ) |
			((x>>TSB)            << (BTSB+BTSB) ) ;
	location += 	((y>>TSB)<<(BTSB+BTSB))*gridDim.x;
		
	int intensityData = intensity[location];
	int     labelData = labels[location];
	
	int sharedX = tx*THREAD_TILE_SIZE+1;
	int sharedY = ty*THREAD_TILE_SIZE+1;
		
	labelTile[sharedY  ][sharedX  ] = labelData         & 0xFF;
	labelTile[sharedY  ][sharedX+1] = (labelData >>  8) & 0xFF;
	labelTile[sharedY+1][sharedX  ] = (labelData >> 16) & 0xFF;
	labelTile[sharedY+1][sharedX+1] = (labelData >> 24) & 0xFF;
	
	intensityTile[sharedY  ][sharedX  ] = intensityData         & 0xFF;
	intensityTile[sharedY  ][sharedX+1] = (intensityData >>  8) & 0xFF;
	intensityTile[sharedY+1][sharedX  ] = (intensityData >> 16) & 0xFF;
	intensityTile[sharedY+1][sharedX+1] = (intensityData >> 24) & 0xFF;

	// Read Border Data into Shared Memory
	/////////////////////////////////////////////////////////////////////////////////////

	// Registers meant for speed. Two given each thread will update 2 pixels.
	int shiftTileReg1 = 0;
	int shiftTileReg2 = 0;
	
	int borderXLoc = 0;
	int borderYLoc = 0;
	
	// Needed Variables
	int bLocation;
	int borderIntData;
	int borderLabData;
	
	// Update horizontal border
	borderXLoc = (tx<<TTSB)+1;
	if (ty == 0 ){		
		// Location to write in shared memory
		borderYLoc = 0;
		if (by != 0) {
			// Upper block border
			y-=THREAD_TILE_SIZE;
			shiftTileReg1 = 16;
			shiftTileReg2 = 24;
		}
	} else if (ty == BLOCK_TILE_SIZE-1){
		// Location to write in shared memory
		borderYLoc = TILE_SIZE+1;			
		if (by != gridDim.y-1) {
			// Lower block border
			y+=THREAD_TILE_SIZE;
			shiftTileReg1 = 0;
			shiftTileReg2 = 8;
		}
	}
	// Read from global and write to shared memory
	if (ty == 0 || ty == BLOCK_TILE_SIZE-1) {
		if ((by == 0           && ty == 0                ) || 
		    (by == gridDim.y-1 && ty == BLOCK_TILE_SIZE-1)){
			intensityTile[borderYLoc][borderXLoc  ] = 0;
			intensityTile[borderYLoc][borderXLoc+1] = 0;
			    labelTile[borderYLoc][borderXLoc  ] = 0;
			    labelTile[borderYLoc][borderXLoc+1] = 0;
			      phiTile[borderYLoc][borderXLoc  ] = 0;
			      phiTile[borderYLoc][borderXLoc+1] = 0;
		} else {
			bLocation = (((x>>TTSB)&BTSMask)                ) |
				    (((y>>TTSB)&BTSMask) << BTSB        ) |
				     ((x>>TSB)            << (BTSB+BTSB) ) ;
			bLocation += ((y>>TSB)<<(BTSB+BTSB))*gridDim.x;
			
			borderIntData = intensity[bLocation];
			borderLabData =    labels[bLocation];
			
			intensityTile[borderYLoc][borderXLoc  ] 
					= ( borderIntData >> shiftTileReg1 ) & 0xFF;
			intensityTile[borderYLoc][borderXLoc+1] 
					= ( borderIntData >> shiftTileReg2 ) & 0xFF;
			    labelTile[borderYLoc][borderXLoc  ] 
					= ( borderLabData >> shiftTileReg1 ) & 0xFF;
			    labelTile[borderYLoc][borderXLoc+1] 
					= ( borderLabData >> shiftTileReg2 ) & 0xFF;
			// Update Phi of border
			if (ty == 0 || ty == BLOCK_TILE_SIZE-1){
				if (labelTile[borderYLoc][borderXLoc] == targetLabel && 
				    intensityTile[borderYLoc][borderXLoc] >= lowerIntensityBound && 
				    intensityTile[borderYLoc][borderXLoc] <= upperIntensityBound){
					phiTile[borderYLoc][borderXLoc] = 1;
				} else if (intensityTile[borderYLoc][borderXLoc] >= lowerIntensityBound && 
					   intensityTile[borderYLoc][borderXLoc] <= upperIntensityBound) {
					phiTile[borderYLoc][borderXLoc] = 2;
				} else {
					phiTile[borderYLoc][borderXLoc] = 0;
				}
				if (labelTile[borderYLoc][borderXLoc+1] == targetLabel && 
				    intensityTile[borderYLoc][borderXLoc+1] >= lowerIntensityBound && 
				    intensityTile[borderYLoc][borderXLoc+1] <= upperIntensityBound){
					phiTile[borderYLoc][borderXLoc+1] = 1;
				} else if (intensityTile[borderYLoc][borderXLoc+1] >= lowerIntensityBound && 
					   intensityTile[borderYLoc][borderXLoc+1] <= upperIntensityBound) {
					phiTile[borderYLoc][borderXLoc+1] = 2;
				} else {
					phiTile[borderYLoc][borderXLoc+1] = 0;
				}
			}
		}
		
	}
	
	// Update vertical border
	x = bx<<BTSB;
	x = x + tx;
	x = x<<TTSB;
	y = by<<BTSB;
	y = y + ty;
	y = y<<TTSB;
	
	borderYLoc = (ty<<TTSB)+1;
	if (tx == 0 ){		
		// Location to write in shared memory
		borderXLoc = 0;
		if (bx != 0) {
			// Upper block border
			x-=THREAD_TILE_SIZE;
			shiftTileReg1 = 8;
			shiftTileReg2 = 24;
		}
	} else if (tx == BLOCK_TILE_SIZE-1){
		// Location to write in shared memory
		borderXLoc = TILE_SIZE+1;			
		if (bx != gridDim.x-1) {
			// Lower block border
			x+=THREAD_TILE_SIZE;
			shiftTileReg1 = 0;
			shiftTileReg2 = 16;
		}
	}
	// Read from global and write to shared memory
	if (tx == 0 || tx == BLOCK_TILE_SIZE-1) {
		if ((bx == 0           && tx == 0                ) || 
		    (bx == gridDim.x-1 && tx == BLOCK_TILE_SIZE-1)){
			intensityTile[borderYLoc  ][borderXLoc] = 0;
			intensityTile[borderYLoc+1][borderXLoc] = 0;
			    labelTile[borderYLoc  ][borderXLoc] = 0;
			    labelTile[borderYLoc+1][borderXLoc] = 0;
			      phiTile[borderYLoc  ][borderXLoc] = 0;
			      phiTile[borderYLoc+1][borderXLoc] = 0;
		} else {
			bLocation = (((x>>TTSB)&BTSMask)                ) |
				    (((y>>TTSB)&BTSMask) << BTSB        ) |
				     ((x>>TSB)            << (BTSB+BTSB) ) ;
			bLocation += ((y>>TSB)<<(BTSB+BTSB))*gridDim.x;
			
			borderIntData = intensity[bLocation];
			borderLabData =    labels[bLocation];
			
			intensityTile[borderYLoc][borderXLoc  ] 
					= ( borderIntData >> shiftTileReg1 ) & 0xFF;
			intensityTile[borderYLoc+1][borderXLoc] 
					= ( borderIntData >> shiftTileReg2 ) & 0xFF;
			    labelTile[borderYLoc][borderXLoc  ] 
					= ( borderLabData >> shiftTileReg1 ) & 0xFF;
			    labelTile[borderYLoc+1][borderXLoc] 
					= ( borderLabData >> shiftTileReg2 ) & 0xFF;
			// Update Phi of border
			if (tx == 0 || tx == BLOCK_TILE_SIZE-1){
				if (labelTile[borderYLoc][borderXLoc] == targetLabel && 
				    intensityTile[borderYLoc][borderXLoc] >= lowerIntensityBound && 
				    intensityTile[borderYLoc][borderXLoc] <= upperIntensityBound){
					phiTile[borderYLoc][borderXLoc] = 1;
				} else if (intensityTile[borderYLoc][borderXLoc] >= lowerIntensityBound && 
					   intensityTile[borderYLoc][borderXLoc] <= upperIntensityBound) {
					phiTile[borderYLoc][borderXLoc] = 2;
				} else {
					phiTile[borderYLoc][borderXLoc] = 0;
				}
				if (labelTile[borderYLoc+1][borderXLoc] == targetLabel && 
				    intensityTile[borderYLoc+1][borderXLoc] >= lowerIntensityBound && 
				    intensityTile[borderYLoc+1][borderXLoc] <= upperIntensityBound){
					phiTile[borderYLoc+1][borderXLoc] = 1;
				} else if (intensityTile[borderYLoc+1][borderXLoc] >= lowerIntensityBound && 
					   intensityTile[borderYLoc+1][borderXLoc] <= upperIntensityBound) {
					phiTile[borderYLoc+1][borderXLoc] = 2;
				} else {
					phiTile[borderYLoc+1][borderXLoc] = 0;
				}
			}
		}
	}	
	localGBI = 0;
	allOnes = 1;
	
	__syncthreads();
	
	// Algorithm 
	/////////////////////////////////////////////////////////////////////////////////////
	
	// Initialization
	for (int tempY = ty+1; tempY <= TILE_SIZE; tempY+=BLOCK_TILE_SIZE ){
		for (int tempX = tx+1; tempX <= TILE_SIZE; tempX+=BLOCK_TILE_SIZE ){
			
			int ownIntData = intensityTile[tempY][tempX];
			if(ownIntData >= lowerIntensityBound && 
			   ownIntData <= upperIntensityBound) {
				localGBI = 1;
				if (labelTile[tempY][tempX] == targetLabel)
					phiTile[tempY][tempX] = 1;
				else {
					allOnes = 0;
					phiTile[tempY][tempX] = 2;
				}
			} else {
				phiTile[tempY][tempX] = 0;
				allOnes = 0;
			}
		}
	}
   
	// Work
	change = 1;
	
	__syncthreads();
	
	while (change){
		__syncthreads();
		change = 0;
		__syncthreads();
		
		for (int tempY = ty+1; tempY <= TILE_SIZE; tempY+=BLOCK_TILE_SIZE ){
			for (int tempX = tx+1; tempX <= TILE_SIZE; tempX+=BLOCK_TILE_SIZE ){
				
				if((phiTile[tempY+1][tempX  ]  == 1 ||
				    phiTile[tempY-1][tempX  ]  == 1 ||
				    phiTile[tempY  ][tempX-1]  == 1 ||
				    phiTile[tempY  ][tempX+1]  == 1 ) && 
				    phiTile[tempY  ][tempX  ]  == 2){
					phiTile  [tempY][tempX] = 1;
					change = 1;
				}
			}
		}
		__syncthreads();
	}
	
	// Write back to main memory
	int phiData1 = phiTile[sharedY  ][sharedX  ] & 0xFF;
        int phiData2 = phiTile[sharedY  ][sharedX+1] & 0xFF;
        int phiData3 = phiTile[sharedY+1][sharedX  ] & 0xFF;
        int phiData4 = phiTile[sharedY+1][sharedX+1] & 0xFF;

        int phiReturnData = phiData1        |
			   (phiData2 << 8 ) |
			   (phiData3 << 16) |
			   (phiData4 << 24);
			
	phi[location] = phiReturnData;
	
	if (tx == 0 && ty == 0){
		if (allOnes){
			globalBlockIndicator[blockId]=0;
		} else {
			globalBlockIndicator[blockId]=localGBI;
		}
	}
}

/*
 * Lss Step 2 from Pseudo Code
 */
 __global__ void lssStep2(signed int* phi, 
			 int* globalBlockIndicator,
			 int* globalFinishedVariable){

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;
	
	int blockId = by*gridDim.x+bx;
	
	// Including border
	__shared__ signed int    phiTile[TILE_SIZE+2][TILE_SIZE+2]; // input/output

	// Global Block Indicator
	__shared__ volatile signed int borderChange;
	__shared__ volatile signed int change;
		
	// Read Global Block Indicator from global memory
	int localGBI = globalBlockIndicator[blockId];
	
	__syncthreads();
	
	if (localGBI) {
		
		// Read Input Data into Shared Memory
		/////////////////////////////////////////////////////////////////////////////////////

		int x = bx<<BTSB;
		x = x + tx;
		x = x<<TTSB;
		int y = by<<BTSB;
		y = y + ty;
		y = y<<TTSB;
		  
		int location = 	(((x>>TTSB)&BTSMask)                ) |
				(((y>>TTSB)&BTSMask) << BTSB        ) |
				((x>>TSB)            << (BTSB+BTSB) ) ;
		location += 	((y>>TSB)<<(BTSB+BTSB))*gridDim.x;
			
		int phiData = phi[location];
		
		int sharedX = tx*THREAD_TILE_SIZE+1;
		int sharedY = ty*THREAD_TILE_SIZE+1;
		
		phiTile[sharedY  ][sharedX  ] = phiData         & 0xFF;
		phiTile[sharedY  ][sharedX+1] = (phiData >>  8) & 0xFF;
		phiTile[sharedY+1][sharedX  ] = (phiData >> 16) & 0xFF;
		phiTile[sharedY+1][sharedX+1] = (phiData >> 24) & 0xFF;

		// Read Border Data into Shared Memory
		/////////////////////////////////////////////////////////////////////////////////////
		
		// Registers meant for speed. Two given each thread will update 2 pixels.
		int shiftTileReg1 = 0;
		int shiftTileReg2 = 0;
		
		int borderXLoc = 0;
		int borderYLoc = 0;
		
		// Needed Variables
		int bLocation;
		int borderPhiData;
		
		// Update horizontal border
		borderXLoc = sharedX;
		if (ty == 0 ){		
			// Location to write in shared memory
			borderYLoc = 0;
			if (by != 0) {
				// Upper block border
				y-=THREAD_TILE_SIZE;
				shiftTileReg1 = 16;
				shiftTileReg2 = 24;
			}
		} else if (ty == BLOCK_TILE_SIZE-1){
			// Location to write in shared memory
			borderYLoc = TILE_SIZE+1;			
			if (by != gridDim.y-1) {
				// Lower block border
				y+=THREAD_TILE_SIZE;
				shiftTileReg1 = 0;
				shiftTileReg2 = 8;
			}
		}
		// Read from global and write to shared memory
		if (ty == 0 || ty == BLOCK_TILE_SIZE-1) {
			if ((by == 0           && ty == 0                ) || 
			    (by == gridDim.y-1 && ty == BLOCK_TILE_SIZE-1)){
				phiTile[borderYLoc][borderXLoc  ] = 0;
				phiTile[borderYLoc][borderXLoc+1] = 0;
			} else {
				bLocation = (((x>>TTSB)&BTSMask)                 ) |
					    (((y>>TTSB)&BTSMask)  << BTSB        ) |
					     ((x>>TSB)            << (BTSB+BTSB) ) ;
				bLocation += ((y>>TSB)<<(BTSB+BTSB))*gridDim.x;
				
				borderPhiData = phi[bLocation];
				
				phiTile[borderYLoc][borderXLoc  ] 
						= ( borderPhiData >> shiftTileReg1 ) & 0xFF;
				phiTile[borderYLoc][borderXLoc+1] 
						= ( borderPhiData >> shiftTileReg2 ) & 0xFF;
			}
		}
			
		// Update vertical border
		x = bx<<BTSB;
		x = x + tx;
		x = x<<TTSB;
		y = by<<BTSB;
		y = y + ty;
		y = y<<TTSB;
		
		borderYLoc = sharedY;
		if (tx == 0 ){		
			// Location to write in shared memory
			borderXLoc = 0;
			if (bx != 0) {
				// Upper block border
				x-=THREAD_TILE_SIZE;
				shiftTileReg1 = 8;
				shiftTileReg2 = 24;
			}
		} else if (tx == BLOCK_TILE_SIZE-1){
			// Location to write in shared memory
			borderXLoc = TILE_SIZE+1;			
			if (bx != gridDim.x-1) {
				// Lower block border
				x+=THREAD_TILE_SIZE;
				shiftTileReg1 = 0;
				shiftTileReg2 = 16;
			}
		}
		// Read from global and write to shared memory
		if (tx == 0 || tx == BLOCK_TILE_SIZE-1) {
			if ((bx == 0           && tx == 0                ) || 
			    (bx == gridDim.x-1 && tx == BLOCK_TILE_SIZE-1)){
				phiTile[borderYLoc][borderXLoc  ] = 0;
				phiTile[borderYLoc+1][borderXLoc] = 0;
			} else {
				bLocation = (((x>>TTSB)&BTSMask)                 ) |
					    (((y>>TTSB)&BTSMask)  << BTSB        ) |
					     ((x>>TSB)            << (BTSB+BTSB) ) ;
				bLocation += ((y>>TSB)<<(BTSB+BTSB))*gridDim.x;
				
				borderPhiData = phi[bLocation];
				
				phiTile[borderYLoc][borderXLoc  ] 
						= ( borderPhiData >> shiftTileReg1 ) & 0xFF;
				phiTile[borderYLoc+1][borderXLoc] 
						= ( borderPhiData >> shiftTileReg2 ) & 0xFF;
			}
		}
		
		borderChange = 0; // Shared variable
		change = 1; // Shared variable
		__syncthreads();
		
		// Algorithm 
		/////////////////////////////////////////////////////////////////////

		while (change){
			__syncthreads();
			change = 0;
			__syncthreads();
			
			for (int tempY = ty+1; tempY <= TILE_SIZE; tempY+=BLOCK_TILE_SIZE ){
				for (int tempX = tx+1; tempX <= TILE_SIZE; tempX+=BLOCK_TILE_SIZE ){
					
					if((phiTile[tempY+1][tempX  ]  == 1 ||
					    phiTile[tempY-1][tempX  ]  == 1 ||
					    phiTile[tempY  ][tempX-1]  == 1 ||
					    phiTile[tempY  ][tempX+1]  == 1 ) && 
					    phiTile[tempY  ][tempX  ]  == 2){
						phiTile  [tempY][tempX] = 1;
						change = 1;
						if (tempX == 1 || tempX == BLOCK_TILE_SIZE || 
						    tempY == 1 || tempY == BLOCK_TILE_SIZE ){
							borderChange = 1;
						}
					}
				}
			}
			__syncthreads();
		}
		
		if (borderChange){
			int phiData1 = phiTile[sharedY  ][sharedX  ] & 0xFF;
			int phiData2 = phiTile[sharedY  ][sharedX+1] & 0xFF;
			int phiData3 = phiTile[sharedY+1][sharedX  ] & 0xFF;
			int phiData4 = phiTile[sharedY+1][sharedX+1] & 0xFF;

			int phiReturnData = phiData1        |
					   (phiData2 << 8 ) |
					   (phiData3 << 16) |
					   (phiData4 << 24);
					
			phi[location] = phiReturnData;

			if (tx == 0 && ty == 0){
				*globalFinishedVariable = 1;
				globalBlockIndicator[blockId]=0;
				__threadfence();
			}
		}
	}
}

/*
 * Lss Step 3 from Pseudo Code
 */
__global__ void lssStep3(signed int* phi,
			 signed int* phiOut) {

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;
				
	// Including border
	__shared__ signed int    phiTile[TILE_SIZE+2][TILE_SIZE+2]; // input
	__shared__ signed int phiOutTile[TILE_SIZE+2][TILE_SIZE+2]; // output

	// Read Input Data into Shared Memory
	/////////////////////////////////////////////////////////////////////////////////////

	int x = bx<<BTSB;
	x = x + tx;
	x = x<<TTSB;
	int y = by<<BTSB;
	y = y + ty;
	y = y<<TTSB;
	  
	int location = 	(((x>>TTSB)&BTSMask)                ) |
			(((y>>TTSB)&BTSMask) << BTSB        ) |
			((x>>TSB)            << (BTSB+BTSB) ) ;
	location += 	((y>>TSB)<<(BTSB+BTSB))*gridDim.x;
		
	int phiData = phi[location];
	
	int sharedX = tx*THREAD_TILE_SIZE+1;
	int sharedY = ty*THREAD_TILE_SIZE+1;
	
	phiTile[sharedY  ][sharedX  ] = phiData         & 0xFF;
	phiTile[sharedY  ][sharedX+1] = (phiData >>  8) & 0xFF;
	phiTile[sharedY+1][sharedX  ] = (phiData >> 16) & 0xFF;
	phiTile[sharedY+1][sharedX+1] = (phiData >> 24) & 0xFF;

	// Read Border Data into Shared Memory
	/////////////////////////////////////////////////////////////////////////////////////
	
	// Registers meant for speed. Two given each thread will update 2 pixels.
	int shiftTileReg1 = 0;
	int shiftTileReg2 = 0;
	
	int borderXLoc = 0;
	int borderYLoc = 0;
	
	// Needed Variables
	int bLocation;
	int borderPhiData;
	
	// Update horizontal border
	borderXLoc = sharedX;
	if (ty == 0 ){		
		// Location to write in shared memory
		borderYLoc = 0;
		if (by != 0) {
			// Upper block border
			y-=THREAD_TILE_SIZE;
			shiftTileReg1 = 16;
			shiftTileReg2 = 24;
		}
	} else if (ty == BLOCK_TILE_SIZE-1){
		// Location to write in shared memory
		borderYLoc = TILE_SIZE+1;			
		if (by != gridDim.y-1) {
			// Lower block border
			y+=THREAD_TILE_SIZE;
			shiftTileReg1 = 0;
			shiftTileReg2 = 8;
		}
	}
	// Read from global and write to shared memory
	if (ty == 0 || ty == BLOCK_TILE_SIZE-1) {
		if ((by == 0           && ty == 0                ) || 
		    (by == gridDim.y-1 && ty == BLOCK_TILE_SIZE-1)){
			phiTile[borderYLoc][borderXLoc  ] = 0;
			phiTile[borderYLoc][borderXLoc+1] = 0;
		} else {
			bLocation = (((x>>TTSB)&BTSMask)                 ) |
				    (((y>>TTSB)&BTSMask)  << BTSB        ) |
				     ((x>>TSB)            << (BTSB+BTSB) ) ;
			bLocation += ((y>>TSB)<<(BTSB+BTSB))*gridDim.x;
			
			borderPhiData = phi[bLocation];
			
			phiTile[borderYLoc][borderXLoc  ] 
					= ( borderPhiData >> shiftTileReg1 ) & 0xFF;
			phiTile[borderYLoc][borderXLoc+1] 
					= ( borderPhiData >> shiftTileReg2 ) & 0xFF;
		}
	}
		
	// Update vertical border
	x = bx<<BTSB;
	x = x + tx;
	x = x<<TTSB;
	y = by<<BTSB;
	y = y + ty;
	y = y<<TTSB;
	
	borderYLoc = sharedY;
	if (tx == 0 ){		
		// Location to write in shared memory
		borderXLoc = 0;
		if (bx != 0) {
			// Upper block border
			x-=THREAD_TILE_SIZE;
			shiftTileReg1 = 8;
			shiftTileReg2 = 24;
		}
	} else if (tx == BLOCK_TILE_SIZE-1){
		// Location to write in shared memory
		borderXLoc = TILE_SIZE+1;			
		if (bx != gridDim.x-1) {
			// Lower block border
			x+=THREAD_TILE_SIZE;
			shiftTileReg1 = 0;
			shiftTileReg2 = 16;
		}
	}
	// Read from global and write to shared memory
	if (tx == 0 || tx == BLOCK_TILE_SIZE-1) {
		if ((bx == 0           && tx == 0                ) || 
		    (bx == gridDim.x-1 && tx == BLOCK_TILE_SIZE-1)){
			phiTile[borderYLoc][borderXLoc  ] = 0;
			phiTile[borderYLoc+1][borderXLoc] = 0;
		} else {
			bLocation = (((x>>TTSB)&BTSMask)                 ) |
				    (((y>>TTSB)&BTSMask)  << BTSB        ) |
				     ((x>>TSB)            << (BTSB+BTSB) ) ;
			bLocation += ((y>>TSB)<<(BTSB+BTSB))*gridDim.x;
			
			borderPhiData = phi[bLocation];
			
			phiTile[borderYLoc][borderXLoc  ] 
					= ( borderPhiData >> shiftTileReg1 ) & 0xFF;
			phiTile[borderYLoc+1][borderXLoc] 
					= ( borderPhiData >> shiftTileReg2 ) & 0xFF;
		}
	}
	
	__syncthreads();
	
	// Algorithm 
	/////////////////////////////////////////////////////////////////////////////////////
	
	for (int tempY = ty+1; tempY <= TILE_SIZE; tempY+=BLOCK_TILE_SIZE ){
		for (int tempX = tx+1; tempX <= TILE_SIZE; tempX+=BLOCK_TILE_SIZE ){
			
			if(phiTile[tempY][tempX] == 1) {
				if(phiTile[tempY+1][tempX]  == 1 &&
				   phiTile[tempY-1][tempX]  == 1 &&
				   phiTile[tempY][tempX+1]  == 1 &&
				   phiTile[tempY][tempX-1]  == 1 ){
					phiOutTile[tempY][tempX] = 0xFD;
				} else 
					phiOutTile[tempY][tempX] = 0xFF;
			} else
				if(phiTile[tempY+1][tempX]  == 1 ||
				   phiTile[tempY-1][tempX]  == 1 ||
				   phiTile[tempY][tempX+1]  == 1 ||
				   phiTile[tempY][tempX-1]  == 1 ){
					phiOutTile[tempY][tempX] = 1;
				} else 
					phiOutTile[tempY][tempX] = 3;
		}
	}
	
	__syncthreads();

	// Write back to main memory
	int phiData1 = phiOutTile[sharedY  ][sharedX  ] & 0xFF;
        int phiData2 = phiOutTile[sharedY  ][sharedX+1] & 0xFF;
        int phiData3 = phiOutTile[sharedY+1][sharedX  ] & 0xFF;
        int phiData4 = phiOutTile[sharedY+1][sharedX+1] & 0xFF;

        int phiReturnData = phiData1        |
			   (phiData2 << 8 ) |
			   (phiData3 << 16) |
			   (phiData4 << 24);
	
	phiOut[location] = phiReturnData;

}

__global__ void evolveContour(unsigned int* intensity, 
			      unsigned int* labels,
			      signed int* phi,
			      signed int* phiOut, 
			      int gridXSize,
			      int gridYSize,
			      int* targetLabels, 
			      int* lowerIntensityBounds, 
			      int* upperIntensityBounds,
			      int max_iterations, 
			      int* globalBlockIndicator,
			      int* globalFinishedVariable,
			      int* totalIterations ) {
        int tid = threadIdx.x;
	
	// Total iterations
	totalIterations = &totalIterations[tid];
	
	// Size in ints
	int size = (gridXSize*gridYSize)<<(BTSB+BTSB);
	
	// New phi pointer for each label.
	phi    = &phi[tid*size];
	phiOut = &phiOut[tid*size];

	globalBlockIndicator = &globalBlockIndicator[tid*gridXSize*gridYSize];

	// Global synchronization variable
	globalFinishedVariable = &globalFinishedVariable[tid];
	
	dim3 dimGrid(gridXSize, gridYSize);
        dim3 dimBlock(BLOCK_TILE_SIZE, BLOCK_TILE_SIZE);
	
	// Initialize phi array
	lssStep1<<<dimGrid, dimBlock>>>(intensity, 
					labels,  
					phi, 
					targetLabels[tid], 
					lowerIntensityBounds[tid], 
					upperIntensityBounds[tid],
					globalBlockIndicator,
					globalFinishedVariable );
	int iterations = 0;
	do {
		iterations++;
		lssStep2<<<dimGrid, dimBlock>>>(phi, 
					globalBlockIndicator,
					globalFinishedVariable );
		hipDeviceSynchronize();
	} while (atomicExch(globalFinishedVariable,0) && (iterations < max_iterations));
	
	lssStep3<<<dimGrid, dimBlock>>>(phi,
					phiOut);
	
	*totalIterations = iterations;
}

signed int *levelSetSegment(unsigned int *intensity, 
			    unsigned int *labels,
			    int height, 
			    int width,
			    int *targetLabels, 
			    int *lowerIntensityBounds,
			    int *upperIntensityBounds,
			    int numLabels){
	
	#if defined(DEBUG)
		printf("Printing input data\n");
		printf("Height: %d\n", height);
		printf("Width: %d\n", width);
		printf("Num Labels: %d\n", numLabels);
		
		for (int i = 0; i < numLabels; i++){
			printf("target label: %d\n", targetLabels[i]);
			printf("lower bound: %d\n", lowerIntensityBounds[i]);
			printf("upper bound: %d\n", upperIntensityBounds[i]);	
		}
	#endif
	
	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*BLOCK_TILE_SIZE*THREAD_TILE_SIZE;
	int YSize = gridYSize*BLOCK_TILE_SIZE*THREAD_TILE_SIZE;
	
	// Both are the same size (CPU/GPU).
	gpu.size = XSize*YSize*sizeof(char);
	
	// Allocate arrays in GPU memory
	#if defined(VERBOSE)
		printf ("Allocating arrays in GPU memory.\n");
	#endif
	
	#if defined(CUDA_TIMING)
		float Ttime;
		TIMER_CREATE(Ttime);
		TIMER_START(Ttime);
	#endif
	
	checkCuda(hipMalloc((void**)&gpu.targetLabels           , numLabels*sizeof(int)));
        checkCuda(hipMalloc((void**)&gpu.lowerIntensityBounds   , numLabels*sizeof(int)));
        checkCuda(hipMalloc((void**)&gpu.upperIntensityBounds   , numLabels*sizeof(int)));
	checkCuda(hipMalloc((void**)&gpu.intensity              , gpu.size));
	checkCuda(hipMalloc((void**)&gpu.labels                 , gpu.size));
	checkCuda(hipMalloc((void**)&gpu.phi                    , numLabels*gpu.size));
	checkCuda(hipMalloc((void**)&gpu.phiOut                 , numLabels*gpu.size));
	checkCuda(hipMalloc((void**)&gpu.globalBlockIndicator   , numLabels*gridXSize*gridYSize*sizeof(int)));
	checkCuda(hipMalloc((void**)&gpu.globalFinishedVariable , numLabels*sizeof(int)));
	checkCuda(hipMalloc((void**)&gpu.totalIterations        , numLabels*sizeof(int)));
	
	// Allocate result array in CPU memory
	gpu.phiOnCpu = new signed int[gpu.size*numLabels/sizeof(int)];
	gpu.totalIterationsOnCpu = new int [numLabels];
	
        checkCuda(hipMemcpy(gpu.targetLabels, 
			targetLabels, 
			numLabels*sizeof(int), 
			hipMemcpyHostToDevice));

        checkCuda(hipMemcpy(gpu.lowerIntensityBounds, 
			lowerIntensityBounds, 
			numLabels*sizeof(int), 
			hipMemcpyHostToDevice));

        checkCuda(hipMemcpy(gpu.upperIntensityBounds, 
			upperIntensityBounds, 
			numLabels*sizeof(int), 
			hipMemcpyHostToDevice));
			
        checkCuda(hipMemcpy(gpu.intensity, 
			intensity, 
			gpu.size, 
			hipMemcpyHostToDevice));
			
        checkCuda(hipMemcpy(gpu.labels, 
			labels, 
			gpu.size, 
			hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());

	#if defined(CUDA_TIMING)
		float Ktime;
		TIMER_CREATE(Ktime);
		TIMER_START(Ktime);
	#endif
	
	#if defined(VERBOSE)
		printf("Running algorithm on GPU.\n");
	#endif
	
	// Launch kernel to begin image segmenation
	evolveContour<<<1, numLabels>>>(gpu.intensity, 
					gpu.labels,
					gpu.phi,
					gpu.phiOut, 
					gridXSize,
					gridYSize, 
					gpu.targetLabels, 
					gpu.lowerIntensityBounds, 
					gpu.upperIntensityBounds,
					max_iterations,
					gpu.globalBlockIndicator,
					gpu.globalFinishedVariable,
					gpu.totalIterations);
	
	checkCuda(hipDeviceSynchronize());

	#if defined(CUDA_TIMING)
		TIMER_END(Ktime);
		printf("Kernel Execution Time: %f ms\n", Ktime);
	#endif
	
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(gpu.phiOnCpu, 
			gpu.phiOut, 
			numLabels*gpu.size, 
			hipMemcpyDeviceToHost));
	
	checkCuda(hipMemcpy(gpu.totalIterationsOnCpu, 
			gpu.totalIterations, 
			numLabels*sizeof(int), 
			hipMemcpyDeviceToHost));
			
	// Free resources and end the program
	checkCuda(hipFree(gpu.intensity));
	checkCuda(hipFree(gpu.labels));
	checkCuda(hipFree(gpu.phi));
	checkCuda(hipFree(gpu.phiOut));
	checkCuda(hipFree(gpu.targetLabels));
	checkCuda(hipFree(gpu.lowerIntensityBounds));
	checkCuda(hipFree(gpu.upperIntensityBounds));
	checkCuda(hipFree(gpu.globalBlockIndicator));
	checkCuda(hipFree(gpu.globalFinishedVariable));
	
	#if defined(CUDA_TIMING)
		TIMER_END(Ttime);
		printf("Total GPU Execution Time: %f ms\n", Ttime);
	#endif
	
	#if defined(VERBOSE)
		for (int i = 0; i < numLabels; i++){
			printf("target label: %d converged in %d iterations.\n", 
					targetLabels[i],
					gpu.totalIterationsOnCpu[i]);	
		}
	#endif
	
	return(gpu.phiOnCpu);

}
